#include "hip/hip_runtime.h"
//
// Created by dev on 8/27/22.
//
#include "../../kernels/inc/matrix.cuh"
#include "../../kernels/inc/render.cuh"
#include "../../kernels/inc/shader_impl.cuh"
#include "../../util/const.h"
#include "zbuffer.h"
#include "zfiller.h"
#include <hip/hip_vector_types.h>

extern __device__ __constant__ mat<4,4> viewport_matrix;
// extern __device__ mat<4,4> projection_matrix;
// extern __device__ mat<4,4> view_matrix;

__device__ void triangle_zbuffer(float3 pts[3], ZBuffer &zbuffer) {
	float2 bboxmin{float(zbuffer.width-1),  float(zbuffer.height-1)};
	float2 bboxmax{0., 0.};
	float2 clamp{float(zbuffer.width-1), float(zbuffer.height-1)};
	for (int i=0; i<3; i++) {
		bboxmin.x = max(0.0f, min(bboxmin.x, pts[i].x));
		bboxmin.y = max(0.0f, min(bboxmin.y, pts[i].y));

		bboxmax.x = min(clamp.x, max(bboxmax.x, pts[i].x));
		bboxmax.y = min(clamp.y, max(bboxmax.y, pts[i].y));
	}


	float3 P{0, 0, 0};
	int cnt = 0;
	for (P.x=floor(bboxmin.x); P.x<=bboxmax.x; P.x++) {
		for (P.y=floor(bboxmin.y); P.y<=bboxmax.y; P.y++) {
			P.z = 0;
			auto bc_screen  = barycentric(pts, P);
			float bc_screen_idx[3] = {bc_screen.x, bc_screen.y, bc_screen.z};
			if (bc_screen.x < 0 || bc_screen.y < 0 || bc_screen.z < 0)
				continue;
			for (int i = 0; i < 3; i++)
				P.z += pts[i].z * bc_screen_idx[i];
			atomicMax(&zbuffer.zbuffer[int(P.x + P.y * zbuffer.width)], P.z);
			cnt++;
			if (cnt > MAX_PIXELS_PER_KERNEL)
				return;
		}
	}
}

__global__ void fill_zbuffer(DrawCallBaseArgs args, ModelArgs model_args, ZBuffer buffer) {
	int position = blockIdx.x * blockDim.x + threadIdx.x;

	auto &model = model_args.model;

	if (position >= model.n_faces)
		return;

	auto &model_matrix = model_args.model_matrix;

	auto face = model.faces[position];
	float3 screen_coords[3];
	float3 world_coords[3];
	float3 look_dir = args.look_at - args.camera_pos;

	mat<4,4> transform_mat = dot(dot(dot(viewport_matrix, args.projection_matrix), model_matrix), args.view_matrix);

	for (int j = 0; j < 3; j++)
	{
		float3 v = model.vertices[at(face, j)];
		screen_coords[j] = m2v(dot(transform_mat, v2m(v)));
		world_coords[j] = v;
	}

	float3 n = cross(world_coords[2] - world_coords[0], world_coords[1] - world_coords[0]);
	n = normalize(n);
	float intensity = dot(n, look_dir);
	if (intensity > 0)
		triangle_zbuffer(screen_coords, buffer);
}

__global__ void set_kernel(ZBuffer buffer, float set_to)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= buffer.width || y >= buffer.height)
		return;
	buffer.zbuffer[x + y * buffer.width] = set_to; // -FLT_MAX is bad, use cam_z
}

void ZFiller::async_zbuf(DrawCallArgs &args, int model_index) {
	auto &model_args = args.models[model_index];
	auto &model = model_args.model;
	auto n_grid = model.n_faces / 32 + 1;
	auto n_block = 32;
	fill_zbuffer<<<n_grid, n_block, 0, stream>>>(args.base, model_args, zbuffer);
}

ZBuffer ZFiller::get_zbuffer() {
	return ZBuffer{.zbuffer=zbuffer.zbuffer, .width=zbuffer.width, .height=zbuffer.height};
}
void ZFiller::async_reset() {
	const dim3 block(16,16);
	const dim3 grid(divUp(zbuffer.width, block.x), divUp(zbuffer.height, block.y));
	set_kernel<<<grid, block, 0, stream>>>(zbuffer, -FLT_MAX); // fixme set with something else!!!!!!!!!!!!!!!!!!!!!!!!!!
}

void ZFiller::resize(int height, int width) {
	zbuffer.create(height, width);
}
void ZFiller::resize(Image &image) {
	resize(image.height, image.width);
}
