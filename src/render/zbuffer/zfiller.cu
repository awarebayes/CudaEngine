#include "hip/hip_runtime.h"
//
// Created by dev on 8/27/22.
//
#include "../../kernels/inc/render.cuh"
#include "../../kernels/inc/shader_impl.cuh"
#include "../../util/const.h"
#include "zbuffer.h"
#include "zfiller.h"
#include <hip/hip_vector_types.h>
#include <glm/glm.hpp>

__device__ void triangle_zbuffer(glm::vec3 pts[3], ZBuffer &zbuffer) {
	glm::vec2 bboxmin{float(zbuffer.width-1),  float(zbuffer.height-1)};
	glm::vec2 bboxmax{0., 0.};
	glm::vec2 clamp{float(zbuffer.width-1), float(zbuffer.height-1)};
	for (int i=0; i<3; i++) {
		bboxmin.x = max(0.0f, min(bboxmin.x, pts[i].x));
		bboxmin.y = max(0.0f, min(bboxmin.y, pts[i].y));

		bboxmax.x = min(clamp.x, max(bboxmax.x, pts[i].x));
		bboxmax.y = min(clamp.y, max(bboxmax.y, pts[i].y));
	}


	glm::vec3 P{0, 0, 0};
	int cnt = 0;
	for (P.x=floor(bboxmin.x); P.x<=bboxmax.x; P.x++) {
		for (P.y=floor(bboxmin.y); P.y<=bboxmax.y; P.y++) {
			P.z = 0;
			auto bc_screen  = barycentric(pts, P);
			float bc_screen_idx[3] = {bc_screen.x, bc_screen.y, bc_screen.z};
			if (bc_screen.x < 0 || bc_screen.y < 0 || bc_screen.z < 0)
				continue;
			for (int i = 0; i < 3; i++)
				P.z += pts[i].z * bc_screen_idx[i];
			atomicMax(&zbuffer.zbuffer[int(P.x + P.y * zbuffer.width)], P.z);
			cnt++;
			if (cnt > MAX_PIXELS_PER_KERNEL)
				return;
		}
	}
}

__global__ void fill_zbuffer(DrawCallBaseArgs args, ModelArgs model_args, ZBuffer buffer) {
	int position = blockIdx.x * blockDim.x + threadIdx.x;

	auto &model = model_args.model;
	if (position >= model.n_faces)
		return;

	glm::vec3 screen_coords[3];
	glm::vec3 look_dir = args.look_at - args.camera_pos;
	auto face = model.faces[position];
	for (int nthvert = 0; nthvert < 3; nthvert++) {
		int index = face[nthvert];
		glm::vec3 v = model.vertices[index];
		auto mv = glm::vec4(v.x, v.y, v.z, 1.0f);
		auto proj = args.projection * (args.view * (model_args.model_matrix * mv));
		if (proj.w < 0)
			return;
		proj.x = (proj.x + 1.0f) * args.screen_size.x / proj.w;
		proj.y = (proj.y + 1.0f) * args.screen_size.y / proj.w;
		proj.z = (proj.z + 1.0f) / proj.w;
		screen_coords[nthvert] = glm::vec3{proj.x, proj.y, proj.z};
	}

	glm::vec3 n = cross(screen_coords[2] - screen_coords[0], screen_coords[1] - screen_coords[0]);
	if (glm::dot(look_dir, {0, 0, 1}) > 0)
		n = -n;
	if (dot(n, look_dir) > 0) {
		triangle_zbuffer(screen_coords, buffer);
	}
}

__global__ void set_kernel(ZBuffer buffer, float set_to)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= buffer.width || y >= buffer.height)
		return;
	buffer.zbuffer[x + y * buffer.width] = set_to; // -FLT_MAX is bad, use cam_z
}

void ZFiller::async_zbuf(DrawCallArgs &args, int model_index) {
	auto &model_args = args.models[model_index];
	auto &model = model_args.model;
	auto n_grid = model.n_faces / 32 + 1;
	auto n_block = 32;
	fill_zbuffer<<<n_grid, n_block, 0, stream>>>(args.base, model_args, zbuffer);
}

ZBuffer ZFiller::get_zbuffer() {
	return ZBuffer{.zbuffer=zbuffer.zbuffer, .width=zbuffer.width, .height=zbuffer.height};
}
void ZFiller::async_reset() {
	const dim3 block(16,16);
	const dim3 grid(divUp(zbuffer.width, block.x), divUp(zbuffer.height, block.y));
	set_kernel<<<grid, block, 0, stream>>>(zbuffer, -FLT_MAX); // fixme set with something else!!!!!!!!!!!!!!!!!!!!!!!!!!
}

void ZFiller::resize(int height, int width) {
	zbuffer.create(height, width);
}
void ZFiller::resize(Image &image) {
	resize(image.height, image.width);
}
