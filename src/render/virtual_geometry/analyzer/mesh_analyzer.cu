#include "hip/hip_runtime.h"
//
// Created by dev on 10/6/22.
//
#include "../../../shader/all.h"
#include "../../../util/const.h"
#include "mesh_analyzer.h"

__device__ int get_subdivision_count(float area, int threshold) {
	float unit = area;
	int count = 1;
	while (unit > threshold) {
		unit /= 4;
		count *= 4;
	}
	return count;
}

template <typename ShaderType>
__global__ void analyze_faces(DrawCallBaseArgs args, ModelDrawCallArgs model_args, const Image image, int threshold, bool *face_mask, int n_faces, int *new_virtual_faces) {
	int position = blockIdx.x * blockDim.x + threadIdx.x;
	auto &model = model_args.model;
	int max_pos = model.n_faces;
	if (position >= max_pos)
		return;

	auto sh = BaseShader<ShaderType>(model, args.light_dir, args.projection, args.view, model_args.model_matrix, args.screen_size, args);
	for (int i = 0; i < 3; i++)
			sh.vertex(position, i, false);

	auto &pts = sh.pts;
	if (pts[0].y==pts[1].y && pts[0].y==pts[2].y) return;

	glm::vec2 bboxmin{float(image.width-1),  float(image.height-1)};
	glm::vec2 bboxmax{0.01, 0.01};
	glm::vec2 clamp{float(image.width-1), float(image.height-1)};
	for (auto &pt : pts) {
		bboxmin.x = max(0.0f, min(bboxmin.x, pt.x));
		bboxmin.y = max(0.0f, min(bboxmin.y, pt.y));

		bboxmax.x = min(clamp.x, max(bboxmax.x, pt.x));
		bboxmax.y = min(clamp.y, max(bboxmax.y, pt.y));
	}


	float area = (bboxmax.x - bboxmin.x) * (bboxmax.y - bboxmin.y);
	if (area > threshold) {
		auto count = get_subdivision_count(area, threshold);
		atomicAdd(new_virtual_faces, count);
		face_mask[position] = true;
	}
}

void MeshAnalyzer::async_analyze_mesh(const DrawCallArgs &args, const Image &image, int model_index)
{
	auto &model_args = args.models[model_index];
	auto &model = model_args.model;
	auto n_grid = model.n_faces / 32 + 1;
	auto n_block = dim3(32);

	if (model.n_faces > capacity) {
		capacity = model.n_faces;
		hipFreeAsync(face_mask, stream);
		hipMallocAsync(&face_mask, sizeof(float) * capacity, stream);
	}

	hipMemsetAsync(face_mask, 0, sizeof(bool) * model.n_faces, stream);
	switch (model.shader)
	{
		case RegisteredShaders::Default:
			analyze_faces<ShaderDefault><<<n_grid, n_block, 0, stream>>>(args.base, model_args, image, area_threshold, face_mask, capacity, new_vfaces_count);
			break;
		case RegisteredShaders::Water:
			analyze_faces<ShaderWater><<<n_grid, n_block, 0, stream>>>(args.base, model_args, image, area_threshold, face_mask, capacity, new_vfaces_count);
			break;
		case RegisteredShaders::VGeom:
			analyze_faces<ShaderVGeom><<<n_grid, n_block, 0, stream>>>(args.base, model_args, image, area_threshold, face_mask, capacity, new_vfaces_count);
			break;
	}
}
MeshAnalyzer::MeshAnalyzer(int capacity_, int &threshold_) : capacity(capacity_), area_threshold(threshold_), Synchronizable() {
	hipMalloc(&face_mask, sizeof(bool) * capacity);
}
MeshAnalyzer::~MeshAnalyzer() {
	hipFree(&face_mask);
}
