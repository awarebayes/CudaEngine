#include "hip/hip_runtime.h"
//
// Created by dev on 10/20/22.
//

#include "../../../shader/all.h"
#include "../../misc/image.cuh"
#include "geometry_upsampler.h"

/*
__global__ void upsample_faces(ModelRef virtual_model, const ModelDrawCallArgs model_args, bool *disabled_original, bool *disabled_virtual, int *index_position) {
	int position = blockIdx.x * blockDim.x + threadIdx.x;
	auto &model = model_args.model;
	int max_pos = model.n_faces;
	if (position >= max_pos)
		return;

	bool is_disabled = disabled_original[position];
	if (!is_disabled)
		return;
	int my_position = atomicAdd(index_position, 1);

	if (my_position * 4 >= virtual_model.n_faces || my_position * 9 >= virtual_model.n_vertices)
	{
		printf("exceed model capacity! my_position: %d, virtual_model.n_faces: %d\n", my_position, virtual_model.n_faces);
		return;
	}

	disabled_virtual[my_position] = false;
	auto &face = model.faces[position];
	for (int i = 0; i < 3; i++) {
		int index = face[i];
		virtual_model.vertices[my_position * 3 + i] = model.vertices[index];
		virtual_model.normals[my_position * 3 + i] = model.normals[index];
	}
	virtual_model.faces[my_position] = {my_position * 3, my_position * 3 + 1, my_position * 3 + 2};
	virtual_model.textures_for_face[my_position] = model.textures_for_face[position];

	auto virtual_index = virtual_model.textures_for_face[my_position];
	auto original_index = model.textures_for_face[position];
	for (int i = 0; i < 3; i++) {
		virtual_model.textures[virtual_index[i]] = model.textures[original_index[i]];
	}
}
*/

__device__ void add_triangle(ModelRef &virtual_model, glm::ivec3 &face, glm::vec3 vertices[3], glm::vec3 normals[3], glm::vec2 textures[3], bool *disabled_virtual, int *index_position)
{
	int my_position = atomicAdd(index_position, 1);

	if (my_position * 4 >= virtual_model.n_faces || my_position * 9 >= virtual_model.n_vertices)
	{
		printf("exceed model capacity! my_position: %d, virtual_model.n_faces: %d\n", my_position, virtual_model.n_faces);
		return;
	}

	disabled_virtual[my_position] = false;
	for (int i = 0; i < 3; i++) {
		virtual_model.vertices[my_position * 3 + i] = vertices[i];
		virtual_model.normals[my_position * 3 + i] = normals[i];
		virtual_model.textures[my_position * 3 + i] = textures[i];
	}

	virtual_model.faces[my_position] = {my_position * 3, my_position * 3 + 1, my_position * 3 + 2};
	virtual_model.textures_for_face[my_position] = { my_position * 3, my_position * 3 + 1, my_position * 3 + 2 };
}

__device__ void upsample(ModelRef &virtual_model, glm::ivec3 &face, glm::vec3 vertices[3], glm::vec3 normals[3], glm::ivec3 &textures_for_face, glm::vec2 *textures, bool *disabled_virtual, int *index_position)
{

}

__global__ void upsample_faces(ModelRef virtual_model, const ModelDrawCallArgs model_args, bool *disabled_original, bool *disabled_virtual, int *index_position) {
	int position = blockIdx.x * blockDim.x + threadIdx.x;
	auto &model = model_args.model;
	int max_pos = model.n_faces;
	if (position >= max_pos)
		return;

	bool is_disabled = disabled_original[position];
	if (!is_disabled)
		return;

	glm::ivec3 face = model.faces[position];
	glm::vec3 vertices[3] = {
		model.vertices[face[0]],
		model.vertices[face[1]],
		model.vertices[face[2]],
	};

	glm::vec3 normals[3] = {
		model.normals[face[0]],
		model.normals[face[1]],
		model.normals[face[2]],
	};
	glm::vec2 textures[3] = {
		model.textures[model.textures_for_face[position][0]],
		model.textures[model.textures_for_face[position][1]],
		model.textures[model.textures_for_face[position][2]],
	};

	add_triangle(virtual_model, face, vertices, normals, textures, disabled_virtual, index_position);
}


void GeometryUpsampler::async_upsample_geometry(const ModelDrawCallArgs &model_args, bool *disabled_faces_for_original, bool *disabled_faces_for_virtual) {
	auto &model = model_args.model;
	auto n_grid = model.n_faces / 32 + 1;
	auto n_block = dim3(32);

	hipMemsetAsync(position, 0, sizeof(int), stream);
	hipMemsetAsync(disabled_faces_for_virtual, 1, sizeof(bool) * virtual_model.n_faces, stream);
	upsample_faces<<<n_grid, n_block, 0, stream>>>(virtual_model, model_args, disabled_faces_for_original, disabled_faces_for_virtual, position);
}

GeometryUpsampler::GeometryUpsampler(ModelRef &virtual_model_, hipStream_t stream_)  : virtual_model(virtual_model_), stream(stream_)
{
	hipMalloc(&position, sizeof(int));
}
GeometryUpsampler::~GeometryUpsampler() {
	hipFree(position);
}
