//
// Created by dev on 7/11/22.
//
#include "stream_manager.h"
#include <hip/hip_runtime_api.h>

StreamManager::StreamManager()
{
	hipStreamCreate(&render);
	hipStreamCreate(&zreset);
}

StreamManager::~StreamManager()
{
	hipStreamDestroy(render);
	hipStreamDestroy(zreset);
}
void StreamManager::prepare_to_render() const {
	checkCudaErrors(hipStreamSynchronize(zreset));
}
