#include "hip/hip_runtime.h"
#include "../../model/inc/model.h"
#include "../../model/inc/pool.h"
#include "../../util/stream_manager.h"
#include "../inc/matrix.cuh"
#include "../inc/render.cuh"
#include "../inc/shader_impl.cuh"
#include "../inc/util.cuh"
#include <ctime>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <hip/hip_vector_types.h>
#include <thrust/fill.h>

__device__ __constant__ mat<4,4> viewport_matrix{};
__device__ mat<4,4> projection_matrix{};
__device__ mat<4,4> view_matrix{};


__device__ void line(Image &image, int x0, int y0, int x1, int y1) {
	bool steep = false;
	if (std::abs(x0-x1)<std::abs(y0-y1)) {
		swap(x0, y0);
		swap(x1, y1);
		steep = true;
	}
	if (x0>x1) {
		swap(x0, x1);
		swap(y0, y1);
	}
	__syncthreads();

	uint color = rgbaFloatToInt(float4{1.0f, 1.0f, 1.0f, 1.0f});

	for (int x=x0; x<=x1; x++) {
		float t = (x-x0)/(float)(x1-x0);
		int y = y0*(1.-t) + y1*t;
		int x_draw = y * steep + x * (1 - steep);
		int y_draw = x * steep + y * (1 - steep);
		image.set(x_draw, y_draw, color);
	}
}


template <typename Tp>
__device__ __forceinline__ float3 barycentric(float3 *pts, Tp P) {
	auto a = float3{float(pts[2].x-pts[0].x), float(pts[1].x-pts[0].x), float(pts[0].x-P.x)};
	auto b = float3{float(pts[2].y-pts[0].y), float(pts[1].y-pts[0].y), float(pts[0].y-P.y)};
	auto u = cross(a, b);
	float flag = abs(u.z) < 1;
	return float3{
	                -1.0f * flag + (1.0f - flag) * (1.f-(u.x+u.y)/u.z),
	                 1.0f * flag + (1.0f - flag) * (u.y/u.z),
	                 1.0f * flag + (1.0f - flag) * (u.x/u.z)
	};
}


__device__ void triangle_zbuffer(float3 pts[3], Image &image) {
	float2 bboxmin{float(image.width-1),  float(image.height-1)};
	float2 bboxmax{0., 0.};
	float2 clamp{float(image.width-1), float(image.height-1)};
	for (int i=0; i<3; i++) {
		bboxmin.x = max(0.0f, min(bboxmin.x, pts[i].x));
		bboxmin.y = max(0.0f, min(bboxmin.y, pts[i].y));

		bboxmax.x = min(clamp.x, max(bboxmax.x, pts[i].x));
		bboxmax.y = min(clamp.y, max(bboxmax.y, pts[i].y));
	}

	float3 P{0, 0, 0};
	for (P.x=floor(bboxmin.x); P.x<=bboxmax.x; P.x++) {
		for (P.y=floor(bboxmin.y); P.y<=bboxmax.y; P.y++) {
			P.z = 0;
			auto bc_screen  = barycentric(pts, P);
			float bc_screen_idx[3] = {bc_screen.x, bc_screen.y, bc_screen.z};
			if (bc_screen.x < 0 || bc_screen.y < 0 || bc_screen.z < 0)
				continue;
			for (int i = 0; i < 3; i++)
				P.z += pts[i].z * bc_screen_idx[i];
			atomicMax(&image.zbuffer[int(P.x + P.y * image.width)], P.z);
		}
	}
}


__device__ void triangle(DrawCallArgs &args, int position, Image &image) {
	auto &model = args.model;
	auto light_dir = args.light_dir;

	mat<4,4> transform_mat = dot(dot(dot(viewport_matrix, projection_matrix), args.model_matrix), view_matrix);
	auto sh = Shader(model, light_dir);
	sh.uniform_M = transform_mat;

	for (int i = 0; i < 3; i++)
		sh.vertex(position, i);

	auto &pts = sh.pts;
	auto &normals = sh.normals;
	auto &textures = sh.textures;

	if (pts[0].y==pts[1].y && pts[0].y==pts[2].y) return;

	float2 bboxmin{float(image.width-1),  float(image.height-1)};
	float2 bboxmax{0., 0.};
	float2 clamp{float(image.width-1), float(image.height-1)};
	for (auto &pt : pts) {
		bboxmin.x = max(0.0f, min(bboxmin.x, pt.x));
		bboxmin.y = max(0.0f, min(bboxmin.y, pt.y));

		bboxmax.x = min(clamp.x, max(bboxmax.x, pt.x));
		bboxmax.y = min(clamp.y, max(bboxmax.y, pt.y));
	}

	float3 P{0, 0, 0};

	for (P.x=floor(bboxmin.x); P.x <= bboxmax.x; P.x++) {
		for (P.y=floor(bboxmin.y); P.y <= bboxmax.y; P.y++) {
			auto bc_screen  = barycentric(pts, P);
			if (bc_screen.x < 0 || bc_screen.y < 0 || bc_screen.z < 0)
				continue;

			P.z = 0;
			for (int i = 0; i < 3; i++)
				P.z += pts[i].z * at(bc_screen, i);

			if (image.zbuffer[int(P.x + P.y* image.width)] == P.z) {
				uint color;
				sh.fragment(bc_screen, color);
				image.set((int)P.x, (int)P.y, color);
			}
		}
	}
}

__global__ void fill_zbuffer(DrawCallArgs args) {
	auto &model = args.model;
	auto &image = args.image;
	int position = blockIdx.x * blockDim.x + threadIdx.x;

	if (position >= model.n_faces)
		return;
	auto face = model.faces[position];
	float3 screen_coords[3];
	float3 world_coords[3];
	float3 look_dir = args.look_at - args.camera_pos;

	mat<4,4> transform_mat = dot(dot(dot(viewport_matrix, projection_matrix), args.model_matrix), view_matrix);

	for (int j = 0; j < 3; j++)
	{
		float3 v = model.vertices[at(face, j)];
		screen_coords[j] = m2v(dot(transform_mat, v2m(v)));
		world_coords[j] = v;
	}

	float3 n = cross(world_coords[2] - world_coords[0], world_coords[1] - world_coords[0]);
	n = normalize(n);
	float intensity = dot(n, look_dir);
	if (intensity > 0)
		triangle_zbuffer(screen_coords, image);
}


__global__ void draw_faces(DrawCallArgs args) {
	auto &model = args.model;
	auto &image = args.image;

	int position = blockIdx.x * blockDim.x + threadIdx.x;
	if (position >= model.n_faces)
		return;
	auto face = model.faces[position];
	float3 world_coords[3];
	auto look_dir = args.look_at - args.camera_pos;
	for (int j = 0; j < 3; j++)
	{
		float3 v = model.vertices[at(face, j)];
		world_coords[j] = v;
	}

	float3 n = cross(world_coords[2] - world_coords[0], world_coords[1] - world_coords[0]);
	n = normalize(n);
	float intensity = dot(n, look_dir);
	if (intensity > 0)
		triangle(args, position, image);
}


void render_init(int width, int height)
{
	int depth = 255;
	mat<4,4> ViewPort = viewport(width/8, height/8, width*3/4, height*3/4, depth);
	hipMemcpyToSymbol(HIP_SYMBOL(
	        viewport_matrix),
	        &ViewPort,
	        sizeof(mat<4,4>)
	        );
}

void update_device_parameters(const DrawCallArgs &args)
{
	mat<4,4> Projection = identity_matrix<4>();

	Projection.at(3, 2) = -1.f / args.camera_pos.z;
	hipMemcpyToSymbol(HIP_SYMBOL(
	        projection_matrix),
	        &Projection,
	        sizeof(mat<4,4>)
	        );

	mat<4,4> View = lookat(args.camera_pos, args.look_at, {0, 1, 0});
	hipMemcpyToSymbol(HIP_SYMBOL(
        view_matrix),
        &View,
        sizeof(mat<4,4>)
        );
}

double main_cuda_launch(const DrawCallArgs &args, StopWatchInterface *timer) {
	auto streams = SingletonCreator<StreamManager>().get();

	// var for kernel computation timing
	// sync host and start kernel computation timer
	double dKernelTime;
	// sync host and start kernel computation timer
	dKernelTime = 0.0;

	clock_t begin = clock();
	sdkResetTimer(&timer);

	update_device_parameters(args);
	streams->prepare_to_render();

	auto &model = args.model;
	auto &image = args.image;
	int n_grid = model.n_faces / 32 + 1;
	int n_block = 32;

	hipMemsetAsync((void *)args.image.pixels, 0, args.image.width * args.image.height * sizeof(uint), streams->render);
	fill_zbuffer<<<n_grid, n_block, 0, streams->render>>>(args);
	draw_faces<<<n_grid, n_block, 0, streams->render>>>(args);

	checkCudaErrors(hipStreamSynchronize(streams->render));
	thrust::fill(thrust::device, image.zbuffer, image.zbuffer + image.width * image.height, -FLT_MAX);
	clock_t end = clock();
	double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;

	printf("%f ms\n", elapsed_secs * 1000);
	dKernelTime = sdkGetTimerValue(&timer);

	return dKernelTime / 1000.;
}