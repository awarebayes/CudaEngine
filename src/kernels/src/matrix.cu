//
// Created by dev on 7/14/22.
//

#include "../inc/matrix.cuh"
#include <hip/hip_vector_types.h>


__device__ __host__ mat<4, 4> viewport(int x, int y, int w, int h, int depth)
{
	mat<4, 4> result = identity_matrix<4>();
	result.at(0, 0) = (float)w / 2.0f;
	result.at(1, 1) = (float)h / 2.0f;
	result.at(2, 2) = (float)depth / 2.0f;
	result.at(0, 3) = (float)x + (float)w / 2.0f;
	result.at(1, 3) = (float)y + (float)h / 2.0f;
	result.at(2, 3) = (float)depth / 2.0f;
	return result;
}

__device__ __host__ void dbg_print(const mat<4, 1> &mat)
{
	printf(
	        "DBG PRINT\n"
	        "%f\n"
	        "%f\n"
	        "%f\n"
	        "%f\n\n",
	        mat.at(0, 0),
	        mat.at(0, 1),
	        mat.at(0, 2),
	        mat.at(0, 3)
	        );
}

__device__ __host__ void dbg_print(const mat<4, 4> &mat)
{
	printf(
		"DBG PRINT\n"
		"%f %f %f %f\n"
		"%f %f %f %f\n"
		"%f %f %f %f\n"
		"%f %f %f %f\n\n",
		mat.at(0, 0), mat.at(0, 1), mat.at(0, 2), mat.at(0, 3),
		mat.at(1, 0), mat.at(1, 1), mat.at(1, 2), mat.at(1, 3),
		mat.at(2, 0), mat.at(2, 1), mat.at(2, 2), mat.at(2, 3),
		mat.at(3, 0), mat.at(3, 1), mat.at(3, 2), mat.at(3, 3)
	);
}
