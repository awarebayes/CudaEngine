#include "hip/hip_runtime.h"
//
// Created by dev on 7/13/22.
//

#include "../inc/util.cuh"



// Euclidean Distance (x, y, d) = exp((|x - y| / d)^2 / 2)
__device__ float euclideanLen(float4 a, float4 b, float d) {
	float mod = (b.x - a.x) * (b.x - a.x) + (b.y - a.y) * (b.y - a.y) +
	            (b.z - a.z) * (b.z - a.z);

	return __expf(-mod / (2.f * d * d));
}


__device__ float4 rgbaIntToFloat(uint c) {
	float4 rgba;
	rgba.x = (c & 0xff) * 0.003921568627f;        //  /255.0f;
	rgba.y = ((c >> 8) & 0xff) * 0.003921568627f; //  /255.0f;
	rgba.z = ((c >> 16) & 0xff) * 0.003921568627f;//  /255.0f;
	rgba.w = ((c >> 24) & 0xff) * 0.003921568627f;//  /255.0f;
	return rgba;
}

__device__ float atomicMax(float* address, float val)
{
	int* address_as_i = (int*) address;
	int old = *address_as_i, assumed;
	do {
		assumed = old;
		old = ::atomicCAS(address_as_i, assumed,
		                  __float_as_int(::fmaxf(val, __int_as_float(assumed))));
	} while (assumed != old);
	return __int_as_float(old);
}
