#include "hip/hip_runtime.h"
//
// Created by dev on 8/27/22.
//
#include "camera.h"
#include <cmath>
#include <imgui.h>

#include "../kernels/inc/matrix.cuh"
#include "hip/hip_vector_types.h"

float3 Camera::get_look_direction() const {
	float yaw = yaw_pitch.x / 180.0f * M_PI;
	float pitch = yaw_pitch.y / 180.0f * M_PI;
	float3 direction;
	direction.x = std::cos(yaw) * std::cos(pitch);
	direction.y = std::sin(pitch);
	direction.z = std::sin(yaw) * std::cos(pitch);


	return direction;
}

void Camera::display_menu() {
	ImGui::Begin("Camera Controls");
	ImGui::SliderFloat2("Camera XY", &position.x, -10, 10);
	ImGui::SliderFloat("Camera Z", &position.z, -100, 100);
	ImGui::SliderFloat2("Look dir", &yaw_pitch.x, -180, 180);
	ImGui::End();
}

mat<4,4> Camera::get_projection_matrix()
{
	mat<4,4> Projection = identity_matrix<4>();
	Projection.at(3, 2) = -1.f / position.z;
	return Projection;
}

mat<4,4> Camera::get_view_matrix()
{
	return lookat(position, position + get_look_direction(), {0, 1, 0});
}
